#include "hip/hip_runtime.h"
//
// Created by mainf on 2024/9/19.
//
extern "C" {
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 1024        // 矩阵大小
#define NB 32         // 块大小
#define NUM_BLOCKS (N / NB)

// 内核函数：执行块LU分解（GETRF）
__global__ void GETRF(float *A, int n, int nb) {
    __shared__ float block[NB][NB];
    const unsigned int tx = threadIdx.x;
    const unsigned int ty = threadIdx.y;

    // 加载当前块到共享内存
    block[ty][tx] = A[blockIdx.y * nb * n + blockIdx.x * nb + ty * n + tx];
    __syncthreads();

    // 进行LU分解（不选主元）
    for (int k = 0; k < nb; ++k) {
        if (tx >= k && ty == k) {
            block[tx][k] /= block[k][k]; // 计算L部分
        }
        __syncthreads();
        if (tx > k && ty > k) {
            block[ty][tx] -= block[ty][k] * block[k][tx]; // 更新U部分
        }
        __syncthreads();
    }

    // 将结果写回全局内存
    A[blockIdx.y * nb * n + blockIdx.x * nb + ty * n + tx] = block[ty][tx];
}

// 内核函数：更新行块（GESSM）
__global__ void GESSM(float *A, float *L, int n, int nb, int k) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col_offset = (k + 1) * nb;

    for (int i = k + 1; i < n / nb; ++i) {
        // 更新行块
        A[(i * nb + ty) * n + col_offset + tx] -= L[ty * n + col_offset + tx] * A[k * nb * n + col_offset + tx];
    }
}

// 内核函数：更新列块（TSTRF）
__global__ void TSTRF(float *A, float *U, int n, int nb, int k) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_offset = (k + 1) * nb;

    for (int i = k + 1; i < n / nb; ++i) {
        // 更新列块
        A[row_offset * n + i * nb + tx] -= U[ty * n + row_offset + tx] * A[row_offset * n + k * nb + tx];
    }
}

// 内核函数：更新剩余矩阵（SSSM）
__global__ void SSSM(float *A, float *L, float *U, int n, int nb, int k) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_offset = (k + 1) * nb;
    int col_offset = (k + 1) * nb;

    for (int i = k + 1; i < n / nb; ++i) {
        for (int j = k + 1; j < n / nb; ++j) {
            // Schur补矩阵更新
            A[(i * nb + ty) * n + (j * nb + tx)] -= L[ty * n + k * nb + tx] * U[k * nb * n + j * nb + tx];
        }
    }
}
}

// 主函数
int main() {
    // 初始化矩阵A
    float *h_A = (float *) malloc(N * N * sizeof(float));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (i == j) {
                h_A[i * N + j] = 2.0f;
            } else {
                h_A[i * N + j] = 1.0f;
            }
        }
    }

    // 分配GPU内存
    float *d_A;
    hipMalloc((void **) &d_A, N * N * sizeof(float));
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);

    // 配置CUDA网格和块
    dim3 threadsPerBlock(NB, NB);
    dim3 numBlocks(NUM_BLOCKS, NUM_BLOCKS);

    // 分块LU分解流程
    for (int k = 0; k < N / NB; ++k) {
        // 1. GETRF: 分解主块
        GETRF<<<dim3(1, 1), threadsPerBlock>>>(d_A, N, NB);
        hipDeviceSynchronize();

        // 2. GESSM: 更新行块
        GESSM<<<dim3(NUM_BLOCKS - (k + 1), 1), threadsPerBlock>>>(d_A, d_A, N, NB, k);
        hipDeviceSynchronize();

        // 3. TSTRF: 更新列块
        TSTRF<<<dim3(1, NUM_BLOCKS - (k + 1)), threadsPerBlock>>>(d_A, d_A, N, NB, k);
        hipDeviceSynchronize();

        // 4. SSSM: 更新剩余块
        SSSM<<<dim3(NUM_BLOCKS - (k + 1), NUM_BLOCKS - (k + 1)), threadsPerBlock>>>(d_A, d_A, d_A, N, NB, k);
        hipDeviceSynchronize();
    }

    // 将结果复制回主机
    hipMemcpy(h_A, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // 输出部分结果
    for (int i = 0; i < min(N, 10); i++) {
        for (int j = 0; j < min(N, 10); j++) {
            printf("%f ", h_A[i * N + j]);
        }
        printf("\n");
    }

    // 释放内存
    free(h_A);
    hipFree(d_A);

    return 0;
}
